#include "hip/hip_runtime.h"
#include "mod_GPU.hpp"

float *getGaussianMatrix(size_t ksize, double sigma);
uchar *getCircleKernel(size_t diameter);

cv::Mat detectObjectInFrameGPU(const cv::Mat &background, cv::Mat frame)
{
    const int height = frame.rows;
    const int width = frame.cols;
    const int numPixels = height * width;
    const size_t ksize = 15;
    const float *gaussianKernel = getGaussianMatrix(ksize, 2.0);
    const uchar threshold = 20;
    const uchar maxval_tresh = 255;
    const int morphologicalCircleDiameter = 15;
    const uchar *circleKernel = getCircleKernel(morphologicalCircleDiameter);

    uchar3 *d_background;
    uchar3 *d_frame;

    uchar *d_bgd;
    uchar *d_input;
    uchar *d_tmp;

    float *d_gaussianKernel;
    uchar *d_circleKernel;

    hipMalloc(&d_background, numPixels * sizeof(uchar3));
    hipMalloc(&d_frame, numPixels * sizeof(uchar3));
    hipMalloc(&d_input, numPixels * sizeof(uchar));
    hipMalloc(&d_bgd, numPixels * sizeof(uchar));
    hipMalloc(&d_gaussianKernel, ksize * ksize * sizeof(float));
    hipMalloc(&d_circleKernel,
               morphologicalCircleDiameter * morphologicalCircleDiameter
                   * sizeof(uchar));
    hipMalloc(&d_tmp, height * width * sizeof(uchar));
    hipMemset(d_tmp, 0, height * width * sizeof(uchar));

    hipMemcpy(d_background, background.ptr<uchar3>(0),
               numPixels * sizeof(uchar3), hipMemcpyHostToDevice);
    hipMemcpy(d_frame, frame.ptr<uchar3>(0), numPixels * sizeof(uchar3),
               hipMemcpyHostToDevice);
    hipMemcpy(d_gaussianKernel, gaussianKernel, ksize * ksize * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_circleKernel, circleKernel,
               morphologicalCircleDiameter * morphologicalCircleDiameter
                   * sizeof(uchar),
               hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    dim3 gridDim(int(ceil((float)width / blockDim.x)),
                 int(ceil((float)height / blockDim.y)));

    grayscaleGPU<<<gridDim, blockDim>>>(d_background, d_bgd, height, width);
    grayscaleGPU<<<gridDim, blockDim>>>(d_frame, d_input, height, width);

    blurGPU<<<gridDim, blockDim>>>(d_bgd, d_bgd, height, width,
                                   d_gaussianKernel, ksize);
    blurGPU<<<gridDim, blockDim>>>(d_input, d_input, height, width,
                                   d_gaussianKernel, ksize);

    diffGPU<<<gridDim, blockDim>>>(d_bgd, d_input, d_input, height, width);

    thresholdGPU<<<gridDim, blockDim>>>(d_input, d_input, height, width,
                                        threshold, maxval_tresh);

    dilateGPU<<<gridDim, blockDim>>>(d_input, d_tmp, height, width,
                                     d_circleKernel, ksize);
    erodeGPU<<<gridDim, blockDim>>>(d_tmp, d_input, height, width,
                                    d_circleKernel, ksize);
    // hipDeviceSynchronize();
    hipMemcpy(d_input, d_tmp, height * width * sizeof(uchar),
               hipMemcpyDeviceToDevice);

    cv::Mat output(cv::Size(width, height), CV_8UC1);
    hipMemcpy(output.ptr<uchar>(0), d_input, numPixels * sizeof(uchar),
               hipMemcpyDeviceToHost);

    hipFree(d_background);
    hipFree(d_frame);
    hipFree(d_input);
    hipFree(d_bgd);
    hipFree(d_gaussianKernel);
    hipFree(d_circleKernel);
    hipFree(d_tmp);

    delete[] gaussianKernel;
    delete[] circleKernel;

    return output;
}