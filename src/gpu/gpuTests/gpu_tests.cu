#include "hip/hip_runtime.h"
#include <criterion/criterion.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>

#include "mod_GPU.hpp"
#include "utils.hpp"

#define CUDA_WARN(XXX)                                                         \
    do                                                                         \
    {                                                                          \
        if (XXX != hipSuccess)                                                \
            std::cerr << "CUDA Error: " << hipGetErrorString(XXX)             \
                      << ", at line " << __LINE__ << std::endl;                \
        hipDeviceSynchronize();                                               \
    } while (0)

template <typename T>
static void assertArrayEqual(T *arr1, T *arr2, int n)
{
    for (int i = 0; i < n; i++)
        cr_assert_eq(arr1[i], arr2[i],
                     "Expected arr1[%d] = %d, got arr2[%d] = %d", i, arr1[i], i,
                     arr2[i]);
}

static __attribute__((unused)) void printMatrix(uchar *mat, int height,
                                                int width)
{
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
            std::cout << std::setfill(' ') << std::setw(3)
                      << static_cast<unsigned>(mat[i * width + j]) << " ";
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

static __attribute__((unused)) void printMatrix(int *mat, int height, int width)
{
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
            std::cout << std::setfill(' ') << std::setw(3) << mat[i * width + j]
                      << " ";
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

Test(check, pass)
{
    cr_assert(1);
}

__global__ void cudaTest(bool *flag)
{
    *flag = true;
}

Test(check, gpu)
{
    bool flag = false;
    bool *d_flag;

    hipMalloc(&d_flag, sizeof(bool));
    hipMemcpy(d_flag, &flag, sizeof(bool), hipMemcpyHostToDevice);

    cudaTest<<<1, 1>>>(d_flag);

    CUDA_WARN(hipDeviceSynchronize());

    hipMemcpy(&flag, d_flag, sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(d_flag);

    cr_assert(flag);
}

Test(morphologicalGPU, dilation)
{
    // clang-format off
    uchar buffer[11 * 11] = {
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0, 255, 255, 255, 255,   0,   0, 255, 255, 255,   0,
          0, 255, 255, 255, 255,   0,   0, 255, 255, 255,   0,
          0, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
          0, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
          0, 255, 255,   0,   0,   0, 255, 255, 255, 255,   0,
          0, 255, 255,   0,   0,   0, 255, 255, 255, 255,   0,
          0, 255, 255,   0,   0,   0, 255, 255, 255, 255,   0,
          0, 255, 255, 255, 255, 255, 255, 255,   0,   0,   0,
          0, 255, 255, 255, 255, 255, 255, 255,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0
    };

    uchar kernel[3 * 3] = {
        1, 1, 1,
        1, 1, 1,
        1, 1, 1
    };

    uchar expected[11 * 11] = {
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255,   0, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255,   0,   0,
        255, 255, 255, 255, 255, 255, 255, 255, 255,   0,   0
    };
    // clang-format on

    uchar *d_input;
    uchar *d_kernel;
    uchar *d_output;

    uchar *output = (uchar *)malloc(11 * 11 * sizeof(uchar));

    hipMalloc(&d_input, 11 * 11 * sizeof(uchar));
    hipMalloc(&d_kernel, 3 * 3 * sizeof(uchar));
    hipMalloc(&d_output, 11 * 11 * sizeof(uchar));
    hipMemcpy(d_input, buffer, 11 * 11 * sizeof(uchar),
               hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, 3 * 3 * sizeof(uchar), hipMemcpyHostToDevice);

    dim3 blockDim(4, 4);
    dim3 gridDim(int(ceil((float)11 / blockDim.x)),
                 int(ceil((float)11 / blockDim.y)));
    dilateGPU<<<gridDim, blockDim>>>(d_input, d_output, 11, 11, d_kernel, 3);

    CUDA_WARN(hipDeviceSynchronize());

    hipMemcpy(output, d_output, 11 * 11 * sizeof(uchar),
               hipMemcpyDeviceToHost);

    assertArrayEqual(expected, output, 11 * 11);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
    free(output);
}

Test(morphologicalGPU, erosion)
{
    // clang-format off
    uchar buffer[13 * 13] = {
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255,   0, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
        255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255
    };

    uchar kernel[3 * 3] = {
        255, 255, 255,
        255, 255, 255,
        255, 255, 255
    };


    uchar expected[13 * 13] = {
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
        0, 255, 255, 255, 255,   0,   0,   0, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255,   0,   0,   0, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
        0, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,   0,
        0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0
    };
    // clang-format on

    uchar *d_input;
    uchar *d_kernel;
    uchar *d_output;

    uchar *output = (uchar *)malloc(13 * 13 * sizeof(uchar));

    hipMalloc(&d_input, 13 * 13 * sizeof(uchar));
    hipMalloc(&d_kernel, 3 * 3 * sizeof(uchar));
    hipMalloc(&d_output, 13 * 13 * sizeof(uchar));
    hipMemcpy(d_input, buffer, 13 * 13 * sizeof(uchar),
               hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, 3 * 3 * sizeof(uchar), hipMemcpyHostToDevice);

    dim3 blockDim(4, 4);
    dim3 gridDim(int(ceil((float)13 / blockDim.x)),
                 int(ceil((float)13 / blockDim.y)));
    erodeGPU<<<gridDim, blockDim>>>(d_input, d_output, 13, 13, d_kernel, 3);

    CUDA_WARN(hipDeviceSynchronize());

    hipMemcpy(output, d_output, 13 * 13 * sizeof(uchar),
               hipMemcpyDeviceToHost);

    assertArrayEqual(expected, output, 13 * 13);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
    free(output);
}

Test(connectedComponents, simple4comps, .timeout = 3)
{
    constexpr int height = 13;
    constexpr int width = 14;
    // clang-format off
    uchar buffer[height * width] = {
          0,   0, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0, 255, 255, 255,   0,   0,   0,   0,   0,   0, 255, 255,   0,   0,
        255, 255, 255, 255, 255,   0,   0,   0,   0,   0, 255, 255,   0,   0,
          0, 255, 255, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0, 255,   0,   0,   0,   0,   0,   0, 255,   0,   0,   0,
          0,   0, 255, 255, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0
    };

    __attribute__ ((unused)) int expected[height * width] = {
          0,   0,   3,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   3,   3,   3,   0,   0,   0,   0,   0,   0,  25,  25,   0,   0,
          3,   3,   3,   3,   3,   0,   0,   0,   0,   0,  25,  25,   0,   0,
          0,   3,   3,   3,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   3,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0, 130,   0,   0,   0,   0,   0,   0, 137,   0,   0,   0,
          0,   0, 130, 130, 130,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0, 130,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0  
    };
    // clang-format on

    // std::cout << "Input:" << std::endl;
    // printMatrix(buffer, height, width);

    uchar *d_input;
    int *d_output;

    int *output = (int *)malloc(height * width * sizeof(int));

    hipMalloc(&d_input, height * width * sizeof(uchar));
    hipMalloc(&d_output, height * width * sizeof(int));
    hipMemcpy(d_input, buffer, height * width * sizeof(uchar),
               hipMemcpyHostToDevice);

    dim3 blockDim(4, 4);
    dim3 gridDim(int(ceil((float)height / blockDim.x)),
                 int(ceil((float)width / blockDim.y)));
    // connectedComponentsGPU(d_input, d_output, height, width, gridDim,
    // blockDim);
    initCCL<<<gridDim, blockDim>>>(d_input, d_output, height, width);
    CUDA_WARN(hipDeviceSynchronize());

    hipMemcpy(output, d_output, height * width * sizeof(int),
               hipMemcpyDeviceToHost);
    // std::cout << "After initCCL:" << std::endl;
    // printMatrix(output, height, width);

    mergeCCL<<<gridDim, blockDim>>>(d_input, d_output, height, width);
    CUDA_WARN(hipDeviceSynchronize());
    hipMemcpy(output, d_output, height * width * sizeof(int),
               hipMemcpyDeviceToHost);
    // std::cout << "After mergeCCL:" << std::endl;
    // printMatrix(output, height, width);

    compressCCL<<<gridDim, blockDim>>>(d_input, d_output, height, width);
    CUDA_WARN(hipDeviceSynchronize());
    hipMemcpy(output, d_output, height * width * sizeof(int),
               hipMemcpyDeviceToHost);
    // std::cout << "After compressCCL (final):" << std::endl;
    // printMatrix(output, height, width);

    // std::cout << "Expected:" << std::endl;
    // printMatrix(expected, height, width);

    assertArrayEqual(expected, output, height * width);

    hipFree(d_input);
    hipFree(d_output);
    free(output);
}

Test(bboxes, fourBboxes, .timeout = 3)
{
    constexpr int height = 13;
    constexpr int width = 14;
    // clang-format off
    uchar buffer[height * width] = {
          0,   0, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0, 255, 255, 255,   0,   0,   0,   0,   0,   0, 255, 255,   0,   0,
        255, 255, 255, 255, 255,   0,   0,   0,   0,   0, 255, 255,   0,   0,
          0, 255, 255, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0, 255,   0,   0,   0,   0,   0,   0, 255,   0,   0,   0,
          0,   0, 255, 255, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0, 255,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0
    };

    __attribute__ ((unused)) int expected[height * width] = {
          0,   0,   3,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   3,   3,   3,   0,   0,   0,   0,   0,   0,  25,  25,   0,   0,
          3,   3,   3,   3,   3,   0,   0,   0,   0,   0,  25,  25,   0,   0,
          0,   3,   3,   3,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   3,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0, 130,   0,   0,   0,   0,   0,   0, 137,   0,   0,   0,
          0,   0, 130, 130, 130,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0, 130,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,
          0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0,   0  
    };
    // clang-format on

    // std::cout << "Input:" << std::endl;
    // printMatrix(buffer, height, width);

    uchar *d_input;
    int *d_output;

    int *output = (int *)malloc(height * width * sizeof(int));

    hipMalloc(&d_input, height * width * sizeof(uchar));
    hipMalloc(&d_output, height * width * sizeof(int));
    hipMemcpy(d_input, buffer, height * width * sizeof(uchar),
               hipMemcpyHostToDevice);

    dim3 blockDim(4, 4);
    dim3 gridDim(int(ceil((float)height / blockDim.x)),
                 int(ceil((float)width / blockDim.y)));
    // connectedComponentsGPU(d_input, d_output, height, width, gridDim,
    // blockDim);
    initCCL<<<gridDim, blockDim>>>(d_input, d_output, height, width);
    CUDA_WARN(hipDeviceSynchronize());

    hipMemcpy(output, d_output, height * width * sizeof(int),
               hipMemcpyDeviceToHost);
    // std::cout << "After initCCL:" << std::endl;
    // printMatrix(output, height, width);

    mergeCCL<<<gridDim, blockDim>>>(d_input, d_output, height, width);
    CUDA_WARN(hipDeviceSynchronize());
    hipMemcpy(output, d_output, height * width * sizeof(int),
               hipMemcpyDeviceToHost);
    // std::cout << "After mergeCCL:" << std::endl;
    // printMatrix(output, height, width);

    compressCCL<<<gridDim, blockDim>>>(d_input, d_output, height, width);
    CUDA_WARN(hipDeviceSynchronize());
    hipMemcpy(output, d_output, height * width * sizeof(int),
               hipMemcpyDeviceToHost);
    // std::cout << "After compressCCL (final):" << std::endl;
    // printMatrix(output, height, width);

    // std::cout << "Expected:" << std::endl;
    // printMatrix(expected, height, width);

    auto bboxes = getBoundingBoxes(output, width, height);

    // for (const auto &box : bboxes)
    // {
    //     std::cout << box << std::endl;
    // }

    cr_assert_eq(bboxes.size(), 4, "Expected bboxes.size() == 4, got %d",
                 bboxes.size());

    hipFree(d_input);
    hipFree(d_output);
    free(output);
}